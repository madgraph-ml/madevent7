#include "device.h"
#include "tensor.h"
#include "../kernels/kernels.h"

using namespace madevent;
using namespace madevent::cuda;
using namespace madevent::kernels;

void* CudaDevice::allocate(std::size_t size) const {
    void* ptr;
    check_error(hipMalloc(&ptr, size));
    return ptr;
}

void CudaDevice::free(void* ptr) const {
    check_error(hipFree(ptr));
}

void CudaDevice::memcpy(void* to, void* from, std::size_t size) const {
    check_error(hipMemcpy(to, from, size, hipMemcpyDefault));
}

void CudaDevice::tensor_copy(const Tensor& source, Tensor& target) const {
    AsyncCudaDevice(hipStreamPerThread).tensor_copy(source, target);
    check_error(hipStreamSynchronize(hipStreamPerThread));
}

void CudaDevice::tensor_zero(Tensor& tensor) const {
    AsyncCudaDevice(hipStreamPerThread).tensor_zero(tensor);
    check_error(hipStreamSynchronize(hipStreamPerThread));
}

void CudaDevice::tensor_add(const Tensor& source, Tensor& target) const {
    AsyncCudaDevice(hipStreamPerThread).tensor_add(source, target);
    check_error(hipStreamSynchronize(hipStreamPerThread));
}

void CudaDevice::tensor_cpu(const Tensor& source, Tensor& target) const {
    check_error(
        hipMemcpy(target.data(), source.data(), source.byte_size(), hipMemcpyDefault)
    );
}

void* AsyncCudaDevice::allocate(std::size_t size) const {
    void* ptr;
    check_error(hipMallocAsync(&ptr, size, _stream));
    return ptr;
}

void AsyncCudaDevice::free(void* ptr) const {
    check_error(hipFreeAsync(ptr, _stream));
}

void AsyncCudaDevice::memcpy(void* to, void* from, std::size_t size) const {
    check_error(hipMemcpyAsync(to, from, size, hipMemcpyDefault, _stream));
}

void AsyncCudaDevice::tensor_copy(const Tensor& source, Tensor& target) const {
    if (source.dtype() == DataType::dt_float && target.dtype() == DataType::dt_float) {
        tensor_foreach_dynamic<kernel_copy<CudaTypes>, 1, 1>(
            {&source}, {&target}, target.size(0), *this
        );
    } else if (source.dtype() == DataType::dt_int && target.dtype() == DataType::dt_int) {
        tensor_foreach_dynamic<kernel_copy_int<CudaTypes>, 1, 1>(
            {&source}, {&target}, target.size(0), *this
        );
    } else {
        throw std::runtime_error("invalid dtype in copy");
    }
}

void AsyncCudaDevice::tensor_zero(Tensor& tensor) const {
    if (tensor.dtype() == DataType::dt_float) {
        tensor_foreach_dynamic<kernel_zero<CudaTypes>, 1, 1>(
            {&tensor}, {&tensor}, tensor.size(0), *this
        );
    } else if (tensor.dtype() == DataType::dt_int) {
        tensor_foreach_dynamic<kernel_zero_int<CudaTypes>, 1, 1>(
            {&tensor}, {&tensor}, tensor.size(0), *this
        );
    } else {
        throw std::runtime_error("invalid dtype in zero");
    }
}

void AsyncCudaDevice::tensor_add(const Tensor& source, Tensor& target) const {
    tensor_foreach_dynamic<kernel_add_inplace<CudaTypes>, 1, 1>(
        {&source}, {&target}, target.size(0), *this
    );
}

void AsyncCudaDevice::tensor_cpu(const Tensor& source, Tensor& target) const {
    check_error(hipMemcpyAsync(
        target.data(), source.data(), source.byte_size(), hipMemcpyDefault, _stream
    ));
}

extern "C" DevicePtr get_device() {
    return &CudaDevice::instance();
}

