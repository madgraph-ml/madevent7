#include "device.h"
#include "tensor.h"
#include "../kernels/kernels.h"

using namespace madevent;
using namespace madevent::cuda;
using namespace madevent::kernels;

void* CudaDevice::allocate(std::size_t size) const {
    void* ptr;
    check_error(hipMalloc(&ptr, size));
    return ptr;
}

void CudaDevice::free(void* ptr) const {
    check_error(hipFree(ptr));
}

void CudaDevice::memcpy(void* to, void* from, std::size_t size) const {
    check_error(hipMemcpy(to, from, size, hipMemcpyDefault));
}

void CudaDevice::tensor_copy(const Tensor& source, Tensor& target) const {
    //TODO: this only accidentally works for types other than double
    tensor_foreach_dynamic<kernel_copy<CudaTypes>, 1, 1>(
        {&source}, {&target}, target.size(0), AsyncCudaDevice(0)
    );
    check_error(hipDeviceSynchronize());
}

void CudaDevice::tensor_zero(Tensor& tensor) const {
    //TODO: this only accidentally works for types other than double
    tensor_foreach_dynamic<kernel_zero<CudaTypes>, 1, 1>(
        {&tensor}, {&tensor}, tensor.size(0), AsyncCudaDevice(0)
    );
    check_error(hipDeviceSynchronize());
}

void CudaDevice::tensor_add(const Tensor& source, Tensor& target) const {
    tensor_foreach_dynamic<kernel_add_inplace<CudaTypes>, 1, 1>(
        {&source}, {&target}, target.size(0), AsyncCudaDevice(0)
    );
}

void CudaDevice::tensor_cpu(const Tensor& source, Tensor& target) const {
    check_error(
        hipMemcpy(target.data(), source.data(), source.byte_size(), hipMemcpyDefault)
    );
}

void* AsyncCudaDevice::allocate(std::size_t size) const {
    void* ptr;
    check_error(hipMallocAsync(&ptr, size, _stream));
    return ptr;
}

void AsyncCudaDevice::free(void* ptr) const {
    check_error(hipFreeAsync(ptr, _stream));
}

void AsyncCudaDevice::memcpy(void* to, void* from, std::size_t size) const {
    check_error(hipMemcpyAsync(to, from, size, hipMemcpyDefault, _stream));
}

void AsyncCudaDevice::tensor_copy(const Tensor& source, Tensor& target) const {
    //TODO: this only accidentally works for types other than double
    tensor_foreach_dynamic<kernel_copy<CudaTypes>, 1, 1>(
        {&source}, {&target}, target.size(0), *this
    );
}

void AsyncCudaDevice::tensor_zero(Tensor& tensor) const {
    //TODO: this only accidentally works for types other than double
    tensor_foreach_dynamic<kernel_zero<CudaTypes>, 1, 1>(
        {&tensor}, {&tensor}, tensor.size(0), *this
    );
}

void AsyncCudaDevice::tensor_add(const Tensor& source, Tensor& target) const {
    tensor_foreach_dynamic<kernel_add_inplace<CudaTypes>, 1, 1>(
        {&source}, {&target}, target.size(0), *this
    );
}

void AsyncCudaDevice::tensor_cpu(const Tensor& source, Tensor& target) const {
    check_error(hipMemcpyAsync(
        target.data(), source.data(), source.byte_size(), hipMemcpyDefault, _stream
    ));
}

extern "C" DevicePtr get_device() {
    return &CudaDevice::instance();
}

